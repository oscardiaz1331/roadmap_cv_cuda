
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdio>

__global__ void addKernel(float* A, float* B, float* C, int vector_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < vector_size) {
        C[i] = A[i] + B[i];
    }
}

__global__ void addKernelFloat4(float4* A, float4* B, float4* C, int vector_size_float4)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < vector_size_float4) {
        C[i].x = A[i].x + B[i].x;
        C[i].y = A[i].y + B[i].y;
        C[i].z = A[i].z + B[i].z;
        C[i].w = A[i].w + B[i].w;
    }
}

int main(){
    int vector_size = static_cast<int>(pow(2.0, 20.0));
    float* A_host = new float[vector_size];
    float* B_host = new float[vector_size];
    float* C_host = new float[vector_size];

    for (int i = 0; i < vector_size; i++) {
        A_host[i] = 1.0f;
        B_host[i] = 2.0f;
    }

    size_t num_bytes = vector_size * sizeof(float);

    float* A_device;
    float* B_device;
    float* C_device;

    hipMalloc((void**)&A_device, num_bytes);
    hipMalloc((void**)&B_device, num_bytes);
    hipMalloc((void**)&C_device, num_bytes);


    hipMemcpy(A_device, A_host, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(B_device, B_host, num_bytes, hipMemcpyHostToDevice);


    int blockSize = 256;
    int numBlocks = (vector_size + blockSize - 1) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    addKernel<<<numBlocks, blockSize>>>(A_device, B_device, C_device, vector_size);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %f ms\n", milliseconds);
    hipMemcpy(C_host, C_device, num_bytes, hipMemcpyDeviceToHost);

    hipFree(A_device);
    hipFree(B_device);
    hipFree(C_device);


    // Implementing a unified memory approach
    // Data is accessible from both host and device
    float* A_unified;
    float* B_unified;
    float* C_unified;

    hipError_t errA = hipMallocManaged((void**)&A_unified, num_bytes);
    hipError_t errB = hipMallocManaged((void**)&B_unified, num_bytes);
    hipError_t errC = hipMallocManaged((void**)&C_unified, num_bytes);

    if (errA != hipSuccess || errB != hipSuccess || errC != hipSuccess) {
        fprintf(stderr, "Failed to allocate unified memory! Error: %s\n", hipGetErrorString(hipGetLastError()));
        return 1;
    }

    // Initialize data on the host (accessible via A_unified, B_unified pointers)
    for (int i = 0; i < vector_size; i++) {
        A_unified[i] = 1.0f;
        B_unified[i] = 2.0f;
    }

    hipEventRecord(start, 0);
    addKernel<<<numBlocks, blockSize>>>(A_unified, B_unified, C_unified, vector_size);
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Unified Memory - Elapsed kernel time: %f ms\n", milliseconds);

    hipFree(A_unified);
    hipFree(B_unified);
    hipFree(C_unified);

    // Using float4 for vector addition
    int vector_size_float4 = vector_size / 4;
    size_t num_bytes_float4 = vector_size_float4 * sizeof(float4);

    float4* A_device_float4;
    float4* B_device_float4;
    float4* C_device_float4;

    hipMalloc((void**)&A_device_float4, num_bytes_float4);
    hipMalloc((void**)&B_device_float4, num_bytes_float4);
    hipMalloc((void**)&C_device_float4, num_bytes_float4);

    hipMemcpy(A_device_float4, A_host, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(B_device_float4, B_host, num_bytes, hipMemcpyHostToDevice);

    hipEventRecord(start, 0); 
    addKernelFloat4<<<numBlocks, blockSize>>>(A_device_float4, B_device_float4, C_device_float4, vector_size_float4);
    hipEventRecord(stop, 0);  
    hipEventSynchronize(stop); 

    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Float4 Explicit Memcpy - Elapsed kernel time: %f ms\n", milliseconds);

    hipMemcpy(C_host, C_device_float4, num_bytes, hipMemcpyDeviceToHost);

    delete[] A_host;
    delete[] B_host;
    delete[] C_host;
    hipFree(A_device_float4);
    hipFree(B_device_float4);
    hipFree(C_device_float4);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}